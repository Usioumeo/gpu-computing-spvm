#include <hipsparse.h>
#include <hipsparse.h>
#include <sys/select.h>
#include <sys/time.h>
extern "C" {
#include "lib.h"
}

#define ROWS (1 << 13)
#define COLS (1 << 13)
#define NNZ (1 << 24)

#define WARMUPS 0
#define REPS 5

int spmv_csr_gpu_cusparse(CSR *csr, unsigned n, float *input_vec,
                          float *output_vec) {
  if (n != csr->ncol) {
    return 1;
  }
  CSR *gpu_csr = copy_csr_to_gpu(csr);

  float *input_vec_gpu, *output_gpu;
  CHECK_CUDA(hipMalloc(&input_vec_gpu, sizeof(float) * csr->ncol));
  CHECK_CUDA(hipMemcpy(input_vec_gpu, input_vec, sizeof(float) * csr->ncol,
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&output_gpu, sizeof(float) * gpu_csr->nrow));

  // now cusparse handling
  //  cuSPARSE handle and descriptors
  hipsparseHandle_t handle;
  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t input_vec_cuda, output_vec_cuda;
  void *dBuffer = nullptr;
  size_t bufferSize = 0;

  hipsparseCreate(&handle);

  // Create sparse matrix A in CSR format
  hipsparseCreateCsr(&matA, csr->nrow, gpu_csr->ncol, gpu_csr->nnz, gpu_csr->row_idx,
                    gpu_csr->col_idx, gpu_csr->val, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

  // Create dense vectors
  hipsparseCreateDnVec(&input_vec_cuda, csr->ncol, input_vec_gpu, HIP_R_32F);
  hipsparseCreateDnVec(&output_vec_cuda, csr->nrow, output_gpu, HIP_R_32F);

  // Prepare parameters for multiplication
  float alpha = 1.0f, beta = 0.0f;

  // Query buffer size for SpMV
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                          matA, input_vec_cuda, &beta, output_vec_cuda,
                          HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
  hipMalloc(&dBuffer, bufferSize);
  TEST_FUNCTION(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                             matA, input_vec_cuda, &beta, output_vec_cuda,
                             HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, dBuffer);
                hipDeviceSynchronize();)

  // end cusparse handling
  CHECK_CUDA(hipMemcpy(output_vec, output_gpu, sizeof(float) * gpu_csr->nrow,
                        hipMemcpyDeviceToHost));
  // Cleanup
  hipsparseDestroyDnVec(input_vec_cuda);
  hipsparseDestroyDnVec(output_vec_cuda);
  hipsparseDestroySpMat(matA);
  hipsparseDestroy(handle);
  CHECK_CUDA(hipFree(input_vec_gpu));
  CHECK_CUDA(hipFree(output_gpu));
  hipFree(dBuffer);
  //hipFree(input_);
  free_csr_gpu(gpu_csr);
  return 0;
}

int main(int argc, char *argv[]) {
  COO *coo = coo_new();
  CSR *csr = csr_new();
  if (argc > 2) {
    printf("Usage: %s <input_file>\n", argv[0]);
    return -1;
  }
  if (argc == 2) {
    FILE *input = fopen(argv[1], "r");
    if (input == NULL) {
      printf("Error opening file: %s\n", argv[1]);
      return -1;
    }
    if (coo_from_file(input, coo) != 0) {
      printf("Error reading COO from file: %s\n", argv[1]);
      fclose(input);
      return -1;
    }
    coo_to_csr(coo, csr);
    write_bin_to_file(csr, "tmp.bin");
  } else {
    // coo_generate_random(coo, ROWS, COLS, NNZ);
    read_bin_to_csr("tmp.bin", csr);
  }

  printf("csr->nrow %u csr->ncol %u csr->nnz %u\n", csr->nrow, csr->ncol,
         csr->nnz);

  float *input = (float *)malloc(sizeof(float) * csr->ncol);
  // hipHostMalloc(&rand_vec_host, sizeof(float)*COLS);
  for (unsigned i = 0; i < csr->ncol; i++) {
    input[i] = (float)(rand() % 2001 - 1000) * 0.001;
  }

  float *output = (float *)malloc(sizeof(float) * csr->nrow * 2);

  // Timed repetitions
  spmv_csr_gpu_cusparse(csr, csr->ncol, input, output);
  spmv_csr(*csr, csr->ncol, input, output + csr->nrow);

  if (relative_error_compare(output, output + csr->nrow, csr->nrow)) {
    printf("Error in the output\n");
    return -1;
  }

  
  hipFree(output);

  return 0;
}
