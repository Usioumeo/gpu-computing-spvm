#include <hipsparse.h>
#include <hipsparse.h>
#include <sys/select.h>
#include <sys/time.h>
extern "C" {
#include "lib.h"
}


int spmv_csr_gpu_cusparse(CSR *csr, unsigned n, float *input_vec,
                          float *output_vec) {
  if (n != csr->ncol) {
    return 1;
  }
  CSR *gpu_csr = copy_csr_to_gpu(csr);

  float *input_vec_gpu, *output_gpu;
  CHECK_CUDA(hipMalloc(&input_vec_gpu, sizeof(float) * csr->ncol));
  CHECK_CUDA(hipMemcpy(input_vec_gpu, input_vec, sizeof(float) * csr->ncol,
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&output_gpu, sizeof(float) * gpu_csr->nrow));

  // now cusparse handling
  //  cuSPARSE handle and descriptors
  hipsparseHandle_t handle;
  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t input_vec_cuda, output_vec_cuda;
  void *dBuffer = nullptr;
  size_t bufferSize = 0;

  hipsparseCreate(&handle);

  // Create sparse matrix A in CSR format
  hipsparseCreateCsr(&matA, csr->nrow, gpu_csr->ncol, gpu_csr->nnz, gpu_csr->row_idx,
                    gpu_csr->col_idx, gpu_csr->val, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

  // Create dense vectors
  hipsparseCreateDnVec(&input_vec_cuda, csr->ncol, input_vec_gpu, HIP_R_32F);
  hipsparseCreateDnVec(&output_vec_cuda, csr->nrow, output_gpu, HIP_R_32F);

  // Prepare parameters for multiplication
  float alpha = 1.0f, beta = 0.0f;

  // Query buffer size for SpMV
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                          matA, input_vec_cuda, &beta, output_vec_cuda,
                          HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
  hipMalloc(&dBuffer, bufferSize);
  TEST_FUNCTION(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                             matA, input_vec_cuda, &beta, output_vec_cuda,
                             HIP_R_32F, HIPSPARSE_SPMV_CSR_ALG1, dBuffer);
                hipDeviceSynchronize();)

  // end cusparse handling
  CHECK_CUDA(hipMemcpy(output_vec, output_gpu, sizeof(float) * gpu_csr->nrow,
                        hipMemcpyDeviceToHost));
  // Cleanup
  hipsparseDestroyDnVec(input_vec_cuda);
  hipsparseDestroyDnVec(output_vec_cuda);
  hipsparseDestroySpMat(matA);
  hipsparseDestroy(handle);
  CHECK_CUDA(hipFree(input_vec_gpu));
  CHECK_CUDA(hipFree(output_gpu));
  hipFree(dBuffer);
  //hipFree(input_);
  free_csr_gpu(gpu_csr);
  return 0;
}

int main(int argc, char *argv[]) {
  CSR *csr = read_from_file(argc, argv);

  printf("csr->nrow %u csr->ncol %u csr->nnz %u\n", csr->nrow, csr->ncol,
         csr->nnz);

  float *input = (float *)malloc(sizeof(float) * csr->ncol);
  // hipHostMalloc(&rand_vec_host, sizeof(float)*COLS);
  for (unsigned i = 0; i < csr->ncol; i++) {
    input[i] = (float)(rand() % 2001 - 1000) * 0.001;
  }

  float *output = (float *)malloc(sizeof(float) * csr->nrow * 2);

  // Timed repetitions
  spmv_csr_gpu_cusparse(csr, csr->ncol, input, output);
  spmv_csr(*csr, csr->ncol, input, output + csr->nrow);

  if (relative_error_compare(output, output + csr->nrow, csr->nrow)) {
    printf("Error in the output\n");
    return -1;
  }

  
  hipFree(output);
  free(csr);
  return 0;
}
