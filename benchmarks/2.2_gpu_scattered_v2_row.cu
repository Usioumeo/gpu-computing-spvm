#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
extern "C" {
#include "lib.h"
}
#include <math.h>
#include <stdio.h>
#include <sys/select.h>
#include <sys/time.h>
#include <unistd.h>


//how many threads per block
#define BLOCK_THREADS (96)

// size of data_block, so how many consecutive elements to process in a single block
#define DATA_BLOCK (384)
#define WRITE_OUT_BLOCKS 8

__device__ inline unsigned normal_upper_bound(const unsigned *__restrict__ arr, int size, unsigned key) {
  unsigned left = 0;
  unsigned right = size;
  while (left + 1 < right) {
    // printf("left %u right %u\n", left, right);
    unsigned mid = (right + left)>>1;
    if (__ldg(arr + mid) <= key)
      left = mid;
    else
      right = mid;
  }
  return left;
}



__global__ void spmv_csr_gpu_kernel_nnz( const float* __restrict__ val, const unsigned * __restrict__ row_idx, const unsigned* __restrict__ col_idx,
                                           const float* __restrict__ input_vec, float *output_vec, unsigned nrow, unsigned ncol, unsigned nnz, unsigned *cuda_extended_row){
  __shared__ float contributions[DATA_BLOCK];

                                            // for (unsigned i = 0; i < csr.nrow; ++i) {
  unsigned block_start = blockIdx.x * DATA_BLOCK;
  unsigned block_end = min(block_start + DATA_BLOCK, nnz);
  ///build the shared memory with the row_idx
  unsigned assigned_end = min(block_start+DATA_BLOCK*(threadIdx.x+1)/BLOCK_THREADS, block_end);

  unsigned start = block_start+ threadIdx.x;
  if (start < block_end) {
    for(unsigned i=start; i<block_end; i+= BLOCK_THREADS) {
        contributions[i-block_start]= val[i] * input_vec[col_idx[i]];
    }

  }
  
  __syncthreads();
 
  //accumulate all contributions and write them in a single atomic operation
  if(threadIdx.x<WRITE_OUT_BLOCKS){
    unsigned assigned_start = block_start+(DATA_BLOCK*threadIdx.x/WRITE_OUT_BLOCKS);
    unsigned assigned_end = min(block_start+(DATA_BLOCK*(threadIdx.x+1)/WRITE_OUT_BLOCKS), block_end);
    float contrib = 0.0;
    unsigned prev_row = cuda_extended_row[block_start];
    bool first = true;
    

    for(unsigned i=assigned_start; i<assigned_end; i++) {
      if (cuda_extended_row[i] != prev_row) {
        if (first) {
          atomicAdd(&output_vec[prev_row], contrib);
        } else {
          first = false;
          output_vec[prev_row] = contrib;
        }
        //atomicAdd(&output_vec[prev_row], contrib);
        contrib = 0.0;
        prev_row = cuda_extended_row[i];
      }
      //contrib += contributions[i];
      contrib+= contributions[i-block_start];
    }
    atomicAdd(&output_vec[prev_row], contrib);
  }

  /*for(int i=start; i<block_end; i+= BLOCK_THREADS) {
      float contribution = val[i] * input_vec[col_idx[i]];
      unsigned local_idx = i - block_start;
      unsigned row = shared_rows_idx[local_idx];
      //atomicAdd(&output_vec[row], contribution);
      //output_vec[row] += contribution;
  }*/
  //atomicAdd(&output_vec[prev_row], cur);
}

void dummy_launcher(CSR *csr, float *input_vec, float *output_vec, unsigned *cuda_extended_row) {
  hipMemset(output_vec, 0, sizeof(float) * csr->nrow);
  unsigned int nblocks = (csr->nnz + DATA_BLOCK - 1) / DATA_BLOCK;
  spmv_csr_gpu_kernel_nnz<<<nblocks, BLOCK_THREADS>>>(
      csr->val, csr->row_idx, csr->col_idx, input_vec, output_vec, csr->nrow,
      csr->ncol, csr->nnz, cuda_extended_row);
  /*hipEvent_t kernel_done;
  hipEventCreate(&kernel_done);
  hipEventRecord(kernel_done);
  while (hipEventQuery(kernel_done) == hipErrorNotReady) {
    usleep(100); // Sleep for 100 microseconds
  }
  hipEventDestroy(kernel_done);*/
  CHECK_CUDA(hipDeviceSynchronize());
}

int spmv_csr_gpu_nnz(CSR *csr, unsigned n, float *input_vec,
                        float *output_vec) {
  if (n != csr->ncol) {
    return 1;
  }
  CSR *gpu_csr = copy_csr_to_gpu(csr);



  float *input_vec_gpu, *output_gpu;
  CHECK_CUDA(hipMalloc(&input_vec_gpu, sizeof(float) * csr->ncol));
  CHECK_CUDA(hipMemcpy(input_vec_gpu, input_vec, sizeof(float) * csr->ncol,
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&output_gpu, sizeof(float) * gpu_csr->nrow));

  //uncompress rows
  unsigned *extended_row = (unsigned*)malloc(sizeof(unsigned) * csr->nnz);
  for(unsigned r=0; r<csr->nrow; r++){
    for(unsigned j=csr->row_idx[r]; j<csr->row_idx[r+1]; j++){
      extended_row[j]=r;
    }
  }
  unsigned *cuda_extended_row;
  CHECK_CUDA(hipMalloc(&cuda_extended_row, sizeof(unsigned) * gpu_csr->nnz));
  CHECK_CUDA(hipMemcpy(cuda_extended_row, extended_row, sizeof(float) * csr->nnz,
                        hipMemcpyHostToDevice));

  TEST_FUNCTION(dummy_launcher(gpu_csr, input_vec_gpu, output_gpu, cuda_extended_row));

  free(extended_row);
  CHECK_CUDA(hipFree(cuda_extended_row));
  CHECK_CUDA(hipMemcpy(output_vec, output_gpu, sizeof(float) * gpu_csr->nrow,
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipFree(input_vec_gpu));
  CHECK_CUDA(hipFree(output_gpu));
  free_csr_gpu(gpu_csr);
  return 0;
}

int main(int argc, char *argv[]) {
  CSR *csr = read_from_file(argc, argv);

  printf("csr->nrow %u csr->ncol %u csr->nnz %u\n", csr->nrow, csr->ncol,
         csr->nnz);

  float *input = (float *)malloc(sizeof(float) * csr->ncol);
  // hipHostMalloc(&rand_vec_host, sizeof(float)*COLS);
  for (unsigned i = 0; i < csr->ncol; i++) {
    input[i] = (float)(rand() % 2001 - 1000) * 0.001;
  }

  float *output = (float *)malloc(sizeof(float) * csr->nrow * 2);

  spmv_csr_gpu_nnz(csr, csr->ncol, input, output); //, tmp
  spmv_csr(*csr, csr->ncol, input, output + csr->nrow);

  if (relative_error_compare(output, output + csr->nrow, csr->nrow)) {
    printf("Error in the output\n");
    return 0;
  }
  
  csr_free(csr);
  free(input);
  free(output);
  printf("test passed\n\n");
  return 0;
}
