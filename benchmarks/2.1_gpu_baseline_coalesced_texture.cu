#include "hip/hip_runtime.h"
#include <cassert>
extern "C" {
#include "lib.h"
}
#include <math.h>
#include <stdio.h>
#include <sys/select.h>
#include <sys/time.h>

#include <stdint.h>

#define BLOCK_SIZE 16
#define ROW_PER_BLOCK 16
__global__ void spmv_csr_gpu_kernel_texture(CSR csr, unsigned n,
                                            hipTextureObject_t input_vec,
                                            float *output_vec) {
  __shared__ float shared_output[ROW_PER_BLOCK];
  for (unsigned i = threadIdx.x; i < ROW_PER_BLOCK; i += BLOCK_SIZE) {
    shared_output[i] = 0.0;
  }

  // unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
  unsigned starting_row = blockIdx.x * ROW_PER_BLOCK;
  unsigned end_row = (blockIdx.x + 1) * ROW_PER_BLOCK < csr.nrow
                         ? (blockIdx.x + 1) * ROW_PER_BLOCK
                         : csr.nrow;
  for (unsigned i = starting_row; i < end_row; i++) {
    float out = 0.0;
    unsigned start = csr.row_idx[i] + threadIdx.x;
    unsigned end = csr.row_idx[i + 1];

    float *val = csr.val + start;
    unsigned *col = csr.col_idx + start;
    float *val_end = csr.val + end;
    unsigned col_val = __ldg(col);
    while (val < val_end) {
      out += *val * tex1Dfetch<float>(input_vec,
                                      col_val); //__ldg(&input_vec[col_val]);

      col += BLOCK_SIZE;
      col_val = __ldg(col);
      val += BLOCK_SIZE;
    }
    atomicAdd(&shared_output[i - starting_row], out);
    //__syncthreads();
    // shared_output[i] += out;
    // output_vec[i] = out;
  }
  __syncthreads();
  for (unsigned i = threadIdx.x + starting_row; i < end_row; i += BLOCK_SIZE) {
    output_vec[i] = shared_output[i - starting_row];
  }
  /*// for (unsigned i = 0; i < csr.nrow; ++i) {
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < csr.nrow) {
    float out = 0.0;
    unsigned start = csr.row_idx[i];
    unsigned end = csr.row_idx[i + 1];

    float *val = csr.val + start;
    unsigned *col = csr.col_idx + start;
    float *val_end = csr.val + end;

    while (val < val_end) {

      out += *val * __ldg(&input_vec[*col]);
      val++;
      col++;
    }
    output_vec[i] = out;
  }

  //}*/
}

void dummy_launcher_texture(CSR *csr, float *input_vec, float *output_vec) {
  unsigned nblocks = (csr->nrow + ROW_PER_BLOCK - 1) / ROW_PER_BLOCK;

  // Create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = input_vec;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = 32; // 32-bit float
  resDesc.res.linear.sizeInBytes = csr->ncol * sizeof(float);

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipTextureObject_t input_tex = 0;
  CHECK_CUDA(hipCreateTextureObject(&input_tex, &resDesc, &texDesc, NULL));

  // Launch kernel with texture
  spmv_csr_gpu_kernel_texture<<<nblocks, BLOCK_SIZE>>>(*csr, csr->ncol,
                                                       input_tex, output_vec);
  CHECK_CUDA(hipDeviceSynchronize());

  // Clean up texture object
  CHECK_CUDA(hipDestroyTextureObject(input_tex));
}

int spmv_csr_gpu(CSR *csr, unsigned n, float *input_vec, float *output_vec) {
  if (n != csr->ncol) {
    return 1;
  }
  CSR *gpu_csr = copy_csr_to_gpu(csr);

  float *input_vec_gpu, *output_gpu;
  CHECK_CUDA(hipMalloc(&input_vec_gpu, sizeof(float) * csr->ncol));
  CHECK_CUDA(hipMemcpy(input_vec_gpu, input_vec, sizeof(float) * csr->ncol,
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&output_gpu, sizeof(float) * gpu_csr->nrow));

  TEST_FUNCTION(dummy_launcher_texture(gpu_csr, input_vec_gpu, output_gpu));

  CHECK_CUDA(hipMemcpy(output_vec, output_gpu, sizeof(float) * gpu_csr->nrow,
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipFree(input_vec_gpu));
  CHECK_CUDA(hipFree(output_gpu));
  free_csr_gpu(gpu_csr);
  return 0;
}

int main(int argc, char *argv[]) {
  CSR *csr = common_read_from_file(argc, argv);

  float *input = common_generate_random_input(csr);

  float *output = (float *)malloc(sizeof(float) * csr->nrow * 2);

  spmv_csr_gpu(csr, csr->ncol, input, output); //, tmp
  spmv_csr(*csr, csr->ncol, input, output + csr->nrow);

  if (relative_error_compare(output, output + csr->nrow, csr->nrow)) {
    printf("Error in the output\n");
    return -1;
  }

  csr_free(csr);
  free(input);
  free(output);
  printf("test passed\n\n");
  return 0;
}