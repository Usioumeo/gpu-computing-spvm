#include "hip/hip_runtime.h"
#include <cassert>
extern "C" {
#define USE_CUDA
#include "lib.h"
}
#include <math.h>
#include <stdio.h>
#include <sys/select.h>
#include <sys/time.h>

#include <stdint.h>  
#define ROWS (1 << 13)
#define COLS (1 << 13)
#define NNZ (1 << 24)

#define WARMUPS 4
#define REPS 10

#define BLOCK_SIZE 16
#define ROW_PER_BLOCK 16
__global__ void spmv_csr_gpu_kernel(CSR csr, unsigned n, float *__restrict__ input_vec,
                                    float *output_vec) {
  __shared__ float shared_output[ROW_PER_BLOCK];
  for(unsigned i=threadIdx.x; i<ROW_PER_BLOCK; i+=BLOCK_SIZE){
    shared_output[i] = 0.0;
  }
  
  //unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
  unsigned starting_row = blockIdx.x*ROW_PER_BLOCK;
  unsigned end_row = (blockIdx.x+1)*ROW_PER_BLOCK<csr.nrow ? (blockIdx.x+1)*ROW_PER_BLOCK : csr.nrow;
  for (unsigned i = starting_row; i<end_row; i++) {
    float out = 0.0;
    unsigned start = csr.row_idx[i]+threadIdx.x;
    unsigned end = csr.row_idx[i + 1];

    float *val = csr.val + start;
    unsigned *col = csr.col_idx + start;
    float *val_end = csr.val + end;
    unsigned col_val=__ldg(col);
    while (val < val_end) {
      out += *val * __ldg(&input_vec[col_val]);
      
      col+= BLOCK_SIZE;
      col_val=__ldg(col);
      val+= BLOCK_SIZE;
      
    }
    atomicAdd(&shared_output[i-starting_row], out);
    //__syncthreads();
    //shared_output[i] += out;
    //output_vec[i] = out;
  }
  __syncthreads();
  for(unsigned i=threadIdx.x+starting_row; i<end_row; i+=BLOCK_SIZE){
    output_vec[i]=shared_output[i-starting_row];
  }
  /*// for (unsigned i = 0; i < csr.nrow; ++i) {
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < csr.nrow) {
    float out = 0.0;
    unsigned start = csr.row_idx[i];
    unsigned end = csr.row_idx[i + 1];

    float *val = csr.val + start;
    unsigned *col = csr.col_idx + start;
    float *val_end = csr.val + end;

    while (val < val_end) {
      
      out += *val * __ldg(&input_vec[*col]);
      val++;
      col++;
    }
    output_vec[i] = out;
  }

  //}*/
}

void dummy_launcher(CSR *csr, float *input_vec, float *output_vec) {
  unsigned nblocks = (csr->nrow + ROW_PER_BLOCK- 1) / ROW_PER_BLOCK;
  spmv_csr_gpu_kernel<<<nblocks, BLOCK_SIZE>>>(*csr, csr->ncol, input_vec,
                                               output_vec);
  CHECK_CUDA(hipDeviceSynchronize());
}

int spmv_csr_gpu(CSR *csr, unsigned n, float *input_vec,
                        float *output_vec) {
  if (n != csr->ncol) {
    return 1;
  }
  CSR *gpu_csr = copy_csr_to_gpu(csr);

  float *input_vec_gpu, *output_gpu;
  CHECK_CUDA(hipMalloc(&input_vec_gpu, sizeof(float) * csr->ncol));
  CHECK_CUDA(hipMemcpy(input_vec_gpu, input_vec, sizeof(float) * csr->ncol,
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&output_gpu, sizeof(float) * gpu_csr->nrow));

  TEST_FUNCTION(dummy_launcher(gpu_csr, input_vec_gpu, output_gpu));

  CHECK_CUDA(hipMemcpy(output_vec, output_gpu, sizeof(float) * gpu_csr->nrow,
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipFree(input_vec_gpu));
  CHECK_CUDA(hipFree(output_gpu));
  free_csr_gpu(gpu_csr);
  return 0;
}

int main(int argc, char *argv[]) {
  COO *coo = coo_new();
  CSR *csr = csr_new();
  if (argc > 2) {
    printf("Usage: %s <input_file>\n", argv[0]);
    return -1;
  }
  if (argc == 2) {
    FILE *input = fopen(argv[1], "r");
    if (input == NULL) {
      printf("Error opening file: %s\n", argv[1]);
      return -1;
    }
    if (coo_from_file(input, coo) != 0) {
      printf("Error reading COO from file: %s\n", argv[1]);
      fclose(input);
      return -1;
    }
    coo_to_csr(coo, csr);
    write_bin_to_file(csr, "tmp.bin");
  } else {
    //coo_generate_random(coo, ROWS, COLS, NNZ);
    read_bin_to_csr("tmp.bin", csr);
  }
  
  
  
  printf("csr->nrow %u csr->ncol %u csr->nnz %u\n", csr->nrow, csr->ncol,
         csr->nnz);

  float *input = (float *)malloc(sizeof(float) * csr->ncol);
  // hipHostMalloc(&rand_vec_host, sizeof(float)*COLS);
  for (unsigned i = 0; i < csr->ncol; i++) {
    input[i] = (float)(rand() % 2001 - 1000) * 0.001;
  }

  float *output = (float *)malloc(sizeof(float) * csr->nrow * 2);

  spmv_csr_gpu(csr, csr->ncol, input, output); //, tmp
  spmv_csr(*csr, csr->ncol, input, output + csr->nrow);

  if (relative_error_compare(output, output + csr->nrow, csr->nrow)) {
    printf("Error in the output\n");
    return -1;
  }

  coo_free(coo);
  csr_free(csr);
  free(input);
  free(output);
  printf("test passed\n\n");
  return 0;
}