#include "hip/hip_runtime.h"
#include <cassert>
extern "C" {
#include "lib.h"
}
#include <math.h>
#include <stdio.h>
#include <sys/select.h>
#include <sys/time.h>

#include <stdint.h>  

#define BLOCK_SIZE 16
#define ROW_PER_BLOCK 16
__global__ void spmv_csr_gpu_kernel(CSR csr, unsigned n, float *__restrict__ input_vec,
                                    float *output_vec) {
  __shared__ float shared_output[ROW_PER_BLOCK];
  for(unsigned i=threadIdx.x; i<ROW_PER_BLOCK; i+=BLOCK_SIZE){
    shared_output[i] = 0.0;
  }
  
  //unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
  unsigned starting_row = blockIdx.x*ROW_PER_BLOCK;
  unsigned end_row = (blockIdx.x+1)*ROW_PER_BLOCK<csr.nrow ? (blockIdx.x+1)*ROW_PER_BLOCK : csr.nrow;
  for (unsigned i = starting_row; i<end_row; i++) {
    float out = 0.0;
    unsigned start = csr.row_idx[i]+threadIdx.x;
    unsigned end = csr.row_idx[i + 1];

    float *val = csr.val + start;
    unsigned *col = csr.col_idx + start;
    float *val_end = csr.val + end;
    unsigned col_val=__ldg(col);
    while (val < val_end) {
      out += *val * __ldg(&input_vec[col_val]);
      
      col+= BLOCK_SIZE;
      col_val=__ldg(col);
      val+= BLOCK_SIZE;
      
    }
    atomicAdd(&shared_output[i-starting_row], out);
    //__syncthreads();
    //shared_output[i] += out;
    //output_vec[i] = out;
  }
  __syncthreads();
  for(unsigned i=threadIdx.x+starting_row; i<end_row; i+=BLOCK_SIZE){
    output_vec[i]=shared_output[i-starting_row];
  }
  /*// for (unsigned i = 0; i < csr.nrow; ++i) {
  unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < csr.nrow) {
    float out = 0.0;
    unsigned start = csr.row_idx[i];
    unsigned end = csr.row_idx[i + 1];

    float *val = csr.val + start;
    unsigned *col = csr.col_idx + start;
    float *val_end = csr.val + end;

    while (val < val_end) {
      
      out += *val * __ldg(&input_vec[*col]);
      val++;
      col++;
    }
    output_vec[i] = out;
  }

  //}*/
}

void dummy_launcher(CSR *csr, float *input_vec, float *output_vec) {
  unsigned nblocks = (csr->nrow + ROW_PER_BLOCK- 1) / ROW_PER_BLOCK;
  spmv_csr_gpu_kernel<<<nblocks, BLOCK_SIZE>>>(*csr, csr->ncol, input_vec,
                                               output_vec);
  CHECK_CUDA(hipDeviceSynchronize());
}

int spmv_csr_gpu(CSR *csr, unsigned n, float *input_vec,
                        float *output_vec) {
  if (n != csr->ncol) {
    return 1;
  }
  CSR *gpu_csr = copy_csr_to_gpu(csr);

  float *input_vec_gpu, *output_gpu;
  CHECK_CUDA(hipMalloc(&input_vec_gpu, sizeof(float) * csr->ncol));
  CHECK_CUDA(hipMemcpy(input_vec_gpu, input_vec, sizeof(float) * csr->ncol,
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&output_gpu, sizeof(float) * gpu_csr->nrow));

  TEST_FUNCTION(dummy_launcher(gpu_csr, input_vec_gpu, output_gpu));

  CHECK_CUDA(hipMemcpy(output_vec, output_gpu, sizeof(float) * gpu_csr->nrow,
                        hipMemcpyDeviceToHost));
  CHECK_CUDA(hipFree(input_vec_gpu));
  CHECK_CUDA(hipFree(output_gpu));
  free_csr_gpu(gpu_csr);
  return 0;
}

int main(int argc, char *argv[]) {
  CSR *csr = read_from_file(argc, argv);
  
  
  
  printf("csr->nrow %u csr->ncol %u csr->nnz %u\n", csr->nrow, csr->ncol,
         csr->nnz);

  float *input = (float *)malloc(sizeof(float) * csr->ncol);
  // hipHostMalloc(&rand_vec_host, sizeof(float)*COLS);
  for (unsigned i = 0; i < csr->ncol; i++) {
    input[i] = (float)(rand() % 2001 - 1000) * 0.001;
  }

  float *output = (float *)malloc(sizeof(float) * csr->nrow * 2);

  spmv_csr_gpu(csr, csr->ncol, input, output); //, tmp
  spmv_csr(*csr, csr->ncol, input, output + csr->nrow);

  if (relative_error_compare(output, output + csr->nrow, csr->nrow)) {
    printf("Error in the output\n");
    return -1;
  }
  csr_free(csr);
  free(input);
  free(output);
  printf("test passed\n\n");
  return 0;
}