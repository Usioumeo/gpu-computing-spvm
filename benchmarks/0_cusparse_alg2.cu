#include <hipsparse.h>
#include <hipsparse.h>
#include <sys/select.h>
#include <sys/time.h>
#define USE_CUDA
extern "C" {
#include "lib.h"
}

#define ROWS (1 << 13)
#define COLS (1 << 13)
#define NNZ (1 << 24)

#define WARMUPS 40
#define REPS 500

int main(int argc, char *argv[]) {
  printf("cusparse baseline alg 2\n");
  COO *coo = coo_new();
  if (argc > 2) {
    printf("Usage: %s <input_file>\n", argv[0]);
    return -1;
  }
  if (argc == 2) {
    FILE *input = fopen(argv[1], "r");
    if (input == NULL) {
      printf("Error opening file: %s\n", argv[1]);
      return -1;
    }
    if (coo_from_file(input, coo) != 0) {
      printf("Error reading COO from file: %s\n", argv[1]);
      fclose(input);
      return -1;
    }
  } else {
    coo_generate_random(coo, ROWS, COLS, NNZ);
  }
  CSR *csr = csr_new();
  coo_to_csr(coo, csr);

  float *rand_vec;// = (float *)malloc(sizeof(float) * csr->ncol);
  hipMallocManaged(&rand_vec, sizeof(float) * csr->ncol);
  float *output; //= (float *)malloc(sizeof(float) * csr->ncol * 2);
  hipMallocManaged(&output, sizeof(float) * csr->nrow * 2);
  for (unsigned i = 0; i < csr->ncol; i++) {
    rand_vec[i] = (float)(rand() % 2001 - 1000) * 0.001;
  }
  // cuSPARSE handle and descriptors
  hipsparseHandle_t handle;
  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t input_vec, output_vec;
  void *dBuffer = nullptr;
  size_t bufferSize = 0;

  hipsparseCreate(&handle);

  // Create sparse matrix A in CSR format
  hipsparseCreateCsr(&matA, csr->nrow, csr->ncol, csr->nnz, csr->row_idx,
                    csr->col_idx, csr->val, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

  // Create dense vectors
  hipsparseCreateDnVec(&input_vec, csr->nrow, rand_vec, HIP_R_32F);
  hipsparseCreateDnVec(&output_vec, csr->ncol, output, HIP_R_32F);

  // Prepare parameters for multiplication
  float alpha = 1.0f, beta = 0.0f;

  // Query buffer size for SpMV
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                          matA, input_vec, &beta, output_vec, HIP_R_32F,
                          HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
  hipMalloc(&dBuffer, bufferSize);

  // Timed repetitions

  TEST_FUNCTION(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                             matA, input_vec, &beta, output_vec, HIP_R_32F,
                             HIPSPARSE_SPMV_CSR_ALG2, dBuffer);
                hipDeviceSynchronize();)
  spmv_csr(*csr,  csr->ncol, rand_vec, output +  csr->nrow);

  if (relative_error_compare(output, output + csr->nrow, csr->nrow)) {
    printf("Error in the output\n");
    return -1;
  }
  // Cleanup
  hipsparseDestroyDnVec(input_vec);
  hipsparseDestroyDnVec(output_vec);
  hipsparseDestroySpMat(matA);
  hipsparseDestroy(handle);
  hipFree(dBuffer);
  hipFree(rand_vec);
  hipFree(output);

  return 0;
}
