#include <hipsparse.h>
#include <hipsparse.h>
#include <sys/select.h>
#include <sys/time.h>
extern "C" {
#include "lib.h"
}



int main(int argc, char *argv[]) {
  printf("cusparse baseline alg 2\n");
  CSR *csr = read_from_file(argc, argv);

  float *rand_vec;// = (float *)malloc(sizeof(float) * csr->ncol);
  hipMallocManaged(&rand_vec, sizeof(float) * csr->ncol);
  float *output; //= (float *)malloc(sizeof(float) * csr->ncol * 2);
  hipMallocManaged(&output, sizeof(float) * csr->nrow * 2);
  for (unsigned i = 0; i < csr->ncol; i++) {
    rand_vec[i] = (float)(rand() % 2001 - 1000) * 0.001;
  }
  // cuSPARSE handle and descriptors
  hipsparseHandle_t handle;
  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t input_vec, output_vec;
  void *dBuffer = nullptr;
  size_t bufferSize = 0;

  hipsparseCreate(&handle);

  // Create sparse matrix A in CSR format
  hipsparseCreateCsr(&matA, csr->nrow, csr->ncol, csr->nnz, csr->row_idx,
                    csr->col_idx, csr->val, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

  // Create dense vectors
  hipsparseCreateDnVec(&input_vec, csr->nrow, rand_vec, HIP_R_32F);
  hipsparseCreateDnVec(&output_vec, csr->ncol, output, HIP_R_32F);

  // Prepare parameters for multiplication
  float alpha = 1.0f, beta = 0.0f;

  // Query buffer size for SpMV
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                          matA, input_vec, &beta, output_vec, HIP_R_32F,
                          HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
  hipMalloc(&dBuffer, bufferSize);

  // Timed repetitions

  TEST_FUNCTION(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
                             matA, input_vec, &beta, output_vec, HIP_R_32F,
                             HIPSPARSE_SPMV_CSR_ALG2, dBuffer);
                hipDeviceSynchronize();)
  spmv_csr(*csr,  csr->ncol, rand_vec, output +  csr->nrow);

  if (relative_error_compare(output, output + csr->nrow, csr->nrow)) {
    printf("Error in the output\n");
    return -1;
  }
  // Cleanup
  hipsparseDestroyDnVec(input_vec);
  hipsparseDestroyDnVec(output_vec);
  hipsparseDestroySpMat(matA);
  hipsparseDestroy(handle);
  hipFree(dBuffer);
  hipFree(rand_vec);
  hipFree(output);
  free(csr);
  return 0;
}
