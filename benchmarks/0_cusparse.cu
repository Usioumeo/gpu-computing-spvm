#include <hipsparse.h>
#include <hipsparse.h>
#include <sys/select.h>
#include <sys/time.h>

extern "C" {
#include "lib.h"
}

#define ROWS (1<<13)
#define COLS (1<<13)
#define NNZ (1<<24)

#define WARMUPS 40
#define REPS 500

int main() {
    COO *coo = coo_new();
  // hipMallocManaged(&coo, sizeof(COO));
  coo_generate_random(coo, ROWS, COLS, NNZ);
  CSR *csr = csr_new();
  coo_to_csr(coo, csr);

  // Assign CSR arrays to device pointers for cuSPARSE

  float *rand_vec; // = (float * )malloc(sizeof(float)*COLS);
  hipMallocManaged(&rand_vec, COLS * sizeof(float));
  float *output; //= (float*)malloc(sizeof(float)*COLS*(REPS+1));
  hipMallocManaged(&output, COLS * 2 * sizeof(float));
  memset(output, 0, sizeof(float) * COLS * 2);
  for (unsigned i = 0; i < COLS; i++) {
    rand_vec[i] = (float)(rand() % 2001 - 1000) * 0.001;
  }

    /*CHECK_CUDA(hipMalloc((void**)&dA_values,     nnz * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dA_columns,    nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&dA_row_offsets,(rows + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&dx,            cols * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&dy,            rows * sizeof(float)));

    CHECK_CUDA(hipMemcpy(dA_values, hA_values, nnz * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_columns, hA_columns, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_row_offsets, hA_row_offsets, (rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dx, hx, cols * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dy, hy, rows * sizeof(float), hipMemcpyHostToDevice));*/

    // cuSPARSE handle and descriptors
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t input_vec, output_vec;
    void* dBuffer = nullptr;
    size_t bufferSize = 0;

    hipsparseCreate(&handle);

    // Create sparse matrix A in CSR format
    hipsparseCreateCsr(&matA, csr->nrow, csr->ncol, csr->nnz,
                                     csr->row_idx, csr->col_idx, csr->val,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // Create dense vectors
    hipsparseCreateDnVec(&input_vec, csr->nrow, rand_vec, HIP_R_32F);
    hipsparseCreateDnVec(&output_vec, csr->ncol, output, HIP_R_32F);

    // Prepare parameters for multiplication
    float alpha = 1.0f, beta = 0.0f;

    // Query buffer size for SpMV
    hipsparseSpMV_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha,
        matA,
        input_vec,
        &beta,
        output_vec,
        HIP_R_32F,
        HIPSPARSE_SPMV_ALG_DEFAULT,
        &bufferSize
    );
    hipMalloc(&dBuffer, bufferSize);


    // Timed repetitions
    
        TEST_FUNCTION(hipsparseSpMV(
            handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha,
            matA,
            input_vec,
            &beta,
            output_vec,
            HIP_R_32F,
            HIPSPARSE_SPMV_ALG_DEFAULT,
            dBuffer
        );
        hipDeviceSynchronize();)
    spmv_csr(*csr, COLS, rand_vec, &output[COLS]);
    // Check results
    if(relative_error_compare(output, output+csr->ncol, csr->ncol)) {
        printf("Error in the output\n");
        return -1;
    }
    // Cleanup
    hipsparseDestroyDnVec(input_vec);
    hipsparseDestroyDnVec(output_vec);
    hipsparseDestroySpMat(matA);
    hipsparseDestroy(handle);
    hipFree(dBuffer);
    hipFree(rand_vec);
    hipFree(output);

    return 0;
}
