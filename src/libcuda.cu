extern "C" {
#define USE_CUDA
#include "lib.h"
CSR *copy_csr_to_gpu(CSR *csr) {
  // Move CSR data to GPU
  float *d_val;
  unsigned *d_col_idx, *d_row_idx;
  CSR *ret = (CSR *)malloc(sizeof(CSR));

  CHECK_CUDA(hipMalloc(&d_val, csr->nnz * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_col_idx, csr->nnz * sizeof(unsigned)));
  CHECK_CUDA(hipMalloc(&d_row_idx, (csr->nrow + 1) * sizeof(unsigned)));

  CHECK_CUDA(hipMemcpy(d_val, csr->val, csr->nnz * sizeof(float),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_col_idx, csr->col_idx, csr->nnz * sizeof(unsigned),
                        hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_row_idx, csr->row_idx,
                        (csr->nrow + 1) * sizeof(unsigned),
                        hipMemcpyHostToDevice));

  ret->val = d_val;
  ret->col_idx = d_col_idx;
  ret->row_idx = d_row_idx;
  ret->ncol = csr->ncol; // Keep the number of columns
  ret->nrow = csr->nrow; // Keep the number of rows
  ret->nnz = csr->nnz;   // Keep the number of non-zero

  return ret;
}

void free_csr_gpu(CSR *csr) {
  // Free GPU memory
  CHECK_CUDA(hipFree(csr->val));
  CHECK_CUDA(hipFree(csr->col_idx));
  CHECK_CUDA(hipFree(csr->row_idx));
  free(csr);
}
}