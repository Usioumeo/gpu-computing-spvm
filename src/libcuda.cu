
extern "C" {
  #include "lib.h"
void csr_reserve(CSR *csr, unsigned nnz, unsigned nrow) {
  if (csr->nnz < nnz || csr->nrow < nrow || csr->val == NULL ||
      csr->col_idx == NULL || csr->row_idx == NULL) {
    // resize csr arrays
    if (csr->row_idx != NULL) {
      hipFree(csr->row_idx);
    }
    hipMallocManaged(&csr->row_idx, (nrow + 1) * sizeof(unsigned));
    if (csr->col_idx != NULL) {
      hipFree(csr->col_idx);
    }
    hipMallocManaged(&csr->col_idx, nnz * sizeof(unsigned));
    if (csr->val != NULL) {
      hipFree(csr->val);
    }
    hipMallocManaged(&csr->val, nnz * sizeof(float));
    //(unsigned *)realloc(csr->row_idx, (nrow + 1) * sizeof(unsigned));
    // csr->col_idx = (unsigned *)realloc(csr->col_idx, nnz * sizeof(unsigned));
    // csr->val = (float *)realloc(csr->val, nnz * sizeof(float));
  }
  csr->nnz = nnz;
  csr->nrow = nrow;
}

// Function to free the memory allocated for CSR matrix
// IT ALSO FREES THE POINTER
void csr_free(CSR *csr) {

  hipFree(csr->row_idx);
  hipFree(csr->col_idx);
  hipFree(csr->val);
  csr->row_idx = NULL;
  csr->col_idx = NULL;
  csr->val = NULL;
  hipFree(csr);
}

// Function to create a new empty CSR matrix
CSR *csr_new() {
  CSR *csr; //= (CSR *)malloc(sizeof(CSR));
  hipMallocManaged(&csr, sizeof(CSR));
  csr->row_idx = NULL;
  csr->col_idx = NULL;
  csr->val = NULL;
  csr->nnz = 0;
  csr->nrow = 0;
  csr->ncol = 0;
  return csr;
}
}